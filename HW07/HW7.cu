#include "hip/hip_runtime.h"
// Name: Kyle Earp
// Not simple Julia Set on the GPU
// nvcc HW7.cu -o temp -lglut -lGL

/*
 What to do:
 This code displays a simple Julia set fractal using the GPU.
 But it only runs on a window of 1024X1024.
 Extend it so that it can run on any given window size.
 Also, color it to your liking. I will judge you on your artistic flare. 
 Don't cut off your ear or anything but make Vincent wish he had, had a GPU.
*/

// Include files
#include <stdio.h>
#include <GL/glut.h>

// Defines
#define MAXMAG 10.0 // If you grow larger than this, we assume that you have escaped.
#define MAXITERATIONS 200 // If you have not escaped after this many attempts, we assume you are not going to escape.
#define A  -0.824	//Real part of C
#define B  -0.1711	//Imaginary part of C

// Global variables
unsigned int WindowWidth;
unsigned int WindowHeight;

float XMin = -2.0;
float XMax =  2.0;
float YMin = -2.0;
float YMax =  2.0;

// Function prototypes
void cudaErrorCheck(const char*, int);
__global__ void colorPixels(float*, float, float, float, float, int);

void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

__global__ void colorPixels(float *pixels, float xMin, float yMin, float dx, float dy, int windowWidth) 
{
	float x,y,mag,tempX;
	int count, id, k;
	
	int maxCount = MAXITERATIONS;
	float maxMag = MAXMAG;
	
	for(id = windowWidth*blockIdx.x + threadIdx.x; id < windowWidth*(blockIdx.x + 1); id += blockDim.x)
	{
		//Getting the offset into the pixel buffer.
		//We need the 3 because each pixel has a red, green, and blue value.
		k = 3*id;

		//Assigning each thread its x and y value of its pixel in each iteration of the for loop.
		x = xMin + (id%windowWidth)*dx;
		y = yMin + blockIdx.x*dy;
		float xy = x - y;
		count = 0;
		mag = sqrt(x*x + y*y);;
		while (mag < maxMag && count < maxCount) 
		{
			//We will be changing the x but we need its old value to find y.	
			tempX = x; 
			x = x*x - y*y + A;
			y = (2.0 * tempX * y) + B;
			mag = sqrt(x*x + y*y);
			count++;
		}
		
		//Setting the color values
		if(count < maxCount) //It escaped
		{
			pixels[k]     = 0.0;
			pixels[k + 1] = 0.0;
			pixels[k + 2] = 0.0;
		}
		else //It Stuck around
		{
			// Assumes that xMin = -xMax and yMin = -yMax
			pixels[k]	 = -xy/(xMin + yMin) + 0.85;
			pixels[k + 1] = -xy/(xMin + yMin) + 0.45;
			pixels[k + 2] = -xy/(xMin + yMin) + 0.5;
		}
	}
}

void display(void) 
{ 
	dim3 blockSize, gridSize;
	float *pixelsCPU, *pixelsGPU; 
	float stepSizeX, stepSizeY;
	
	//We need the 3 because each pixel has a red, green, and blue value.
	pixelsCPU = (float *)malloc(WindowWidth*WindowHeight*3*sizeof(float));
	hipMalloc(&pixelsGPU,WindowWidth*WindowHeight*3*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	
	stepSizeX = (XMax - XMin)/((float)WindowWidth);
	stepSizeY = (YMax - YMin)/((float)WindowHeight);
	
	//Threads in a block - Constant 1024
	blockSize.x = 1024; //WindowWidth;
	blockSize.y = 1;
	blockSize.z = 1;
	
	//Blocks in a grid - WindowHeight, which has a maximum of 65535
	gridSize.x = WindowHeight;
	gridSize.y = 1;
	gridSize.z = 1;
	
	colorPixels<<<gridSize, blockSize>>>(pixelsGPU, XMin, YMin, stepSizeX, stepSizeY, WindowWidth);
	cudaErrorCheck(__FILE__, __LINE__);
	
	//Copying the pixels that we just colored back to the CPU.
	hipMemcpyAsync(pixelsCPU, pixelsGPU, WindowWidth*WindowHeight*3*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Waiting for the GPU to finish.
	hipDeviceSynchronize();

	//Putting pixels on the screen.
	glDrawPixels(WindowWidth, WindowHeight, GL_RGB, GL_FLOAT, pixelsCPU);
	glFlush();

	//Cleaning up
	free(pixelsCPU);
	hipFree(pixelsGPU);
}

// Add reshape function
void reshape(int w, int h)
{
    WindowWidth = w;
    WindowHeight = h;
    glViewport(0, 0, w, h);
}

int main(int argc, char** argv)
{ 
   	glutInit(&argc, argv);
	
	// Get the screen width and height
	WindowWidth = glutGet(GLUT_SCREEN_WIDTH);
	WindowHeight = glutGet(GLUT_SCREEN_HEIGHT);
	
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(WindowWidth, WindowHeight);
	glutCreateWindow("Fractals--Man--Fractals");
   	glutDisplayFunc(display);
	// Add reshape function
	glutReshapeFunc(reshape);
   	glutMainLoop();
}
