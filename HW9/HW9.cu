// Name: Kyle Earp
// Vector Dot product on many block 
// nvcc HW9.cu -o temp
/*
 What to do:
 This code is the solution to HW8. It finds the dot product of vectors that are smaller than the block size.
 Extend this code so that it uses many blocks and many threads and can find the dot product of any vector length.
 Use shared memory in your blocks to speed up your code.
 You will have to do the final reduction on the CPU.
 Set your thread count to 200. Set N to different values to check your code.
*/

// Include files

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

// Defines
#define N 20367 // Length of the vector
#define BLOCK_SIZE 200 // Number of threads in a block

// Global variables
float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers
float DotCPU, DotGPU;
dim3 BlockSize; //This variable will hold the Dimensions of your blocks
dim3 GridSize; //This variable will hold the Dimensions of your grid
float Tolerance = 0.01;

// Function prototypes
void cudaErrorCheck(const char *, int);
void setUpDevices();
void allocateMemory();
void initialize();
void dotProductCPU(float*, float*, float*, int);
__global__ void dotProductGPU(float*, float*, float*, int);
bool  check(float, float, float);
long elaspedTime(struct timeval, struct timeval);
void cleanUp();

// This check to see if an error happened in your CUDA code. It tell you what it thinks went wrong,
// and what file and line it occured on.
void cudaErrorCheck(const char *file, int line)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: message = %s, File = %s, Line = %d\n", hipGetErrorString(error), file, line);
		exit(0);
	}
}

// This will be the layout of the parallel space we will be using.
void setUpDevices()
{
	BlockSize.x = BLOCK_SIZE;
	BlockSize.y = 1;
	BlockSize.z = 1;
	
	GridSize.x = (int)((N-1)/BlockSize.x) + 1;
	GridSize.y = 1;
	GridSize.z = 1;

	// Hard coding the check for the block size.
	if(1024 < BlockSize.x)
	{
		printf("\n\n The block size is too large. It must be less than 1024\n");
		exit(0);
	}

	// Hard coding the check for the grid size.
	if(65535 < GridSize.x)
	{
		printf("\n\n The grid size is too large. It must be less than 65535\n");
		exit(0);
	}
}

// Allocating the memory we will be using.
void allocateMemory()
{	
	// Host "CPU" memory.				
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
	
	// Device "GPU" Memory
	hipMalloc(&A_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&B_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
	hipMalloc(&C_GPU,N*sizeof(float));
	cudaErrorCheck(__FILE__, __LINE__);
}

// Loading values into the vectors that we will add.
void initialize()
{
	for(int i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)(3*i);
	}
}

// Adding vectors a and b on the CPU then stores result in vector c.
void dotProductCPU(float *a, float *b, float *C_CPU, int n)
{
	for(int id = 0; id < n; id++)
	{ 
		C_CPU[id] = a[id] * b[id];
	}
	
	for(int id = 1; id < n; id++)
	{
		C_CPU[0] += C_CPU[id];
	}
}

// This is the kernel. It is the function that will run on the GPU.
// It adds vectors a and b on the GPU then stores result in vector c.
__global__ void dotProductGPU(float *a, float *b, float *c, int n)
{
	// Shared memory for the block
	__shared__ float temp[BLOCK_SIZE];

	// The id of the thread in relation to the entire parallel structure.
	int id = threadIdx.x + blockDim.x*blockIdx.x;
	
	// Setting all the values in the shared memory to zero.
	temp[threadIdx.x] = 0.0;

	// Multiplying the components of the vectors together if the id is less than n and store them into shared memory
	if(id < n)
	{
		temp[threadIdx.x] = a[id] * b[id];
	}
	
	// Making sure all the threads are at the same place.
	__syncthreads();
	
	// Doing the reduction on the shared memory.
	int fold = blockDim.x;
	while(1 < fold)
	{
		if(fold%2 != 0)
		{
			if(threadIdx.x == 0 && (id + fold - 1) < n)
			{
				temp[0] += temp[fold - 1];
			}
			fold--;
		}
		fold /= 2;
		if(threadIdx.x < fold && (id + fold) < n)
		{
			temp[threadIdx.x] += temp[threadIdx.x + fold];
		}
		__syncthreads();
	}

	// Putting the answer into the global memory.
	if(threadIdx.x == 0)
	{
		c[id] = temp[0];
	}
}

// Checking to see if anything went wrong in the vector addition.
bool check(float cpuAnswer, float gpuAnswer, float tolerence)
{
	double percentError;
	
	percentError = abs((gpuAnswer - cpuAnswer)/(cpuAnswer))*100.0;
	printf("\n\n percent error = %lf\n", percentError);
	
	if(percentError < Tolerance) 
	{
		return(true);
	}
	else 
	{
		return(false);
	}
}

// Calculating elasped time.
long elaspedTime(struct timeval start, struct timeval end)
{
	// tv_sec = number of seconds past the Unix epoch 01/01/1970
	// tv_usec = number of microseconds past the current second.
	
	long startTime = start.tv_sec * 1000000 + start.tv_usec; // In microseconds.
	long endTime = end.tv_sec * 1000000 + end.tv_usec; // In microseconds

	// Returning the total time elasped in microseconds
	return endTime - startTime;
}

// Cleaning up memory after we are finished.
void cleanUp()
{
	// Freeing host "CPU" memory.
	free(A_CPU); 
	free(B_CPU); 
	free(C_CPU);
	
	hipFree(A_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(B_GPU); 
	cudaErrorCheck(__FILE__, __LINE__);
	hipFree(C_GPU);
	cudaErrorCheck(__FILE__, __LINE__);
}

int main()
{
	timeval start, end;
	long timeCPU, timeGPU;
	//float localC_CPU, localC_GPU;
	
	// Setting up the GPU
	setUpDevices();
	
	// Allocating the memory you will need.
	allocateMemory();
	
	// Putting values in the vectors.
	initialize();
	
	// Multiplying on the CPU
	gettimeofday(&start, NULL);
	dotProductCPU(A_CPU, B_CPU, C_CPU, N);
	DotCPU = C_CPU[0];
	gettimeofday(&end, NULL);
	timeCPU = elaspedTime(start, end);
	
	// Multiplying on the GPU
	gettimeofday(&start, NULL);
	
	// Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	cudaErrorCheck(__FILE__, __LINE__);
	
	dotProductGPU<<<GridSize,BlockSize>>>(A_GPU, B_GPU, C_GPU, N);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Copy Memory from GPU to CPU
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	cudaErrorCheck(__FILE__, __LINE__);
	
	// Making sure the GPU and CPU wiat until each other are at the same place.
	hipDeviceSynchronize();
	cudaErrorCheck(__FILE__, __LINE__);

	DotGPU = 0.0; // Do final reduction on the CPU
	for(int i = 0; i < N; i+=BlockSize.x)
	{
		DotGPU += C_CPU[i];
	}

	gettimeofday(&end, NULL);
	timeGPU = elaspedTime(start, end);

	// Checking to see if all went correctly.
	if(check(DotCPU, DotGPU, Tolerance) == false)
	{
		printf("\n\n Something went wrong in the GPU dot product.\n");
	}
	else
	{
		printf("\n\n You did a dot product correctly on the GPU");
		printf("\n The time it took on the CPU was %ld microseconds", timeCPU);
		printf("\n The time it took on the GPU was %ld microseconds", timeGPU);
	}
	
	// Your done so cleanup your room.	
	cleanUp();	
	
	// Making sure it flushes out anything in the print buffer.
	printf("\n\n");
	
	return(0);
}